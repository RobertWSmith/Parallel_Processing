#include "hip/hip_runtime.h"
//
// GPU Julia set application from Sanders and Kandrot (p. 54)
//

#include <stdio.h>
//#include <cutil.h>
#include "cpu_bitmap.h"

#define DIM 1000

struct hipComplex {
   float r;
   float i;
   __device__ hipComplex( float a, float b) : r(a), i(b) {}
   __device__ float magnitude1(void) {
      return r + i;
   }
   __device__ float magnitude2(void) {
      return r*r + i*i;
   }
   __device__ float magnitude3(void) {
      return r*r*r + i*i*i;
   }
   __device__ hipComplex operator*(const hipComplex& a) {
      return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
   }
   __device__ hipComplex operator+(const hipComplex& a) {
      return hipComplex(r+a.r, i+a.i);
   }
   __device__ float braunl() { 
      return magnitude3() * magnitude1();
   }
};

__device__ int julia(int x, int y) {
   const float scale = 1.5;
   float jx = scale * (float) (DIM / 2 - x) / (DIM / 2);
   float jy = scale * (float) (DIM / 2 - y) / (DIM / 2);

   hipComplex c(-0.8, 0.156); // #1 julia
//   hipComplex c(-0.6, 0); // #2 julia2
//   hipComplex c(-0.123, 0.745); // #3 Douady's rabbit fractal - julia3
//   hipComplex c(-0.391, -0.587); // #2 Siegel's disk fractal - julia4
   hipComplex a(jx, jy);

   int i = 0;
   for (i = 0; i < 200; i++) {
      a = a * a + c;
      if (a.magnitude2() > 4) // original
//      if (a.magnitude3() > 1000) // #2
//      if (a.braunl() > 1000) // #3 Braunl function
         return 0;
   }

   return (int) a.magnitude2()%8;
}

__device__ int julia_value(int offset, int value) {
    if (value == 0) {
        if (offset * 4 + 0 == 0 || 
            offset * 4 + 0 == 2 ||
            offset * 4 + 0 == 3 ||
            offset * 4 + 0 == 5 ) {
            return 255;
        } else { return 0; }
    } else if (value == 1) {
        if (offset * 4 + 1 == 0 || 
            offset * 4 + 1 == 1 ||
            offset * 4 + 1 == 3 ||
            offset * 4 + 1 == 7 ) {
            return 255;
        } else { return 0; }
    } else if (value == 2) {
        if (offset * 4 + 2 == 0 || 
            offset * 4 + 2 == 1 ||
            offset * 4 + 2 == 5 ||
            offset * 4 + 2 == 6 ) {
            return 255;
        } else { return 0; }
    }
    return 0;
}

__global__ void kernel(unsigned char *ptr) {
   // map from threadIdx/blockIdx to pixel position
   int x = blockIdx.x;
   int y = blockIdx.y;
   int offset = x + y * gridDim.x;

   // now calculate the value at that position
   int juliaValue = julia(x, y);
   ptr[offset * 4 + 0] = julia_value(juliaValue, 0);
   ptr[offset * 4 + 1] = julia_value(juliaValue, 1);
   ptr[offset * 4 + 2] = julia_value(juliaValue, 2);
   ptr[offset * 4 + 3] = 255;
}

int main(void) {
   CPUBitmap bitmap(DIM, DIM);
   unsigned char *dev_bitmap;

   hipMalloc((void**)&dev_bitmap, bitmap.image_size());
   dim3 grid(DIM, DIM);
   kernel<<<grid,1>>>(dev_bitmap);
   hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
   bitmap.display_and_exit();
   hipFree(dev_bitmap);
}
